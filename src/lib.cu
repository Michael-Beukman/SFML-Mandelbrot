
#include <hip/hip_runtime.h>
#include <stdio.h>

// World Coordinates
__device__ void get_world_coords(int screenx, int screeny, double& worldx, double& worldy,
    double scalex, double scaley, double offsetx, double offsety
){
    worldx = screenx / scalex + offsetx;
    worldy = screeny / scaley + offsety;
}


// This does the julia iteration count
__global__ void get_julia_iters(int* iteration_count, int _WIDTH, int _HEIGHT, int MAX_ITERS, double scalex, double scaley, double offsetx, double offsety)
{

    // get the current thread's x and y values
    int tx = threadIdx.x + blockDim.x * blockIdx.x;
    int ty = threadIdx.y + blockDim.y * blockIdx.y;


    // transform into world coords
    double za, zb;
    get_world_coords(tx, ty, za, zb, scalex, scaley, offsetx, offsety);
    double ca = -0.8, cb = 0.156;

    // loop over
    int iters = 0;
    for (; iters < MAX_ITERS; ++iters) {

        double tempa = za * za - zb * zb + ca;
        double tempb = 2 * za * zb + cb;
        za = tempa; zb = tempb;
        if (za*za + zb*zb >= 4) break;
    }
    // add the number of iterations to the array
    iteration_count[ty * _WIDTH + tx] = iters;
}

// see comments above for julia
__global__ void get_mandelbrot_iters(int* iteration_count, int _WIDTH, int _HEIGHT, int MAX_ITERS, double scalex, double scaley, double offsetx, double offsety)
{
    int tx = threadIdx.x + blockDim.x * blockIdx.x;
    int ty = threadIdx.y + blockDim.y * blockIdx.y;

    double ca, cb;
    get_world_coords(tx, ty, ca, cb, scalex, scaley, offsetx, offsety);

    int iters = 0;
    double za = 0, zb = 0;
    for (; iters < MAX_ITERS; ++iters) {

        double tempa = za * za - zb * zb + ca;
        double tempb = 2 * za * zb + cb;
        za = tempa; zb = tempb;
        if (za*za + zb*zb >= 4) break;
    }
    
    iteration_count[ty * _WIDTH + tx] = iters;
}